#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "utils.h"

#include <stdio.h>
#include <vector>
#include <chrono>

inline constexpr unsigned get_index_from_number(unsigned number)
{
	return (number - 3) / 2;//will never truncate any odd number which is all we work with here.
}

inline constexpr unsigned get_number_from_index(unsigned index)
{
	return (index * 2) + 3;
}

constexpr unsigned maxNumber = 1000000000;
constexpr unsigned arraySize = (maxNumber / 2) - 1;//all evens excluded as well as 1
constexpr unsigned maxRoot = static_cast<unsigned>(utils::ct_sqrt(maxNumber)) + 1;//ensure it rounds up
constexpr unsigned maxRootIndex = get_index_from_number(maxRoot);

__global__ void sieve(unsigned* composites)
{
	unsigned const rootIndex = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned const root = ((rootIndex) * 2) + 3;
	if(rootIndex <= maxRootIndex)//for extra threads if tasks%threads != 0
	{
		#pragma unroll
		for(unsigned curIndex = rootIndex + root; curIndex < arraySize; curIndex += root)
		{
			composites[curIndex] |= ~0;
		}
	}
}

void printResults(long long const duration, unsigned* const composites)
{
	unsigned totalPrimes = 1;//2 is prime but never considered here
	for(unsigned i = 0; i < arraySize; ++i)
	{
		if(!composites[i])
		{
			//std::cout << get_number_from_index(i) << "\n";
			++totalPrimes;
		}
	}
	std::cout << "Found  " << totalPrimes << " primes in " << duration << " nanoseconds, " << duration / (1000 * 1000) << " miliseconds, or " << duration / (1000 * 1000 * 1000) << " seconds.";
}


int main()
{
	auto start(std::chrono::high_resolution_clock::now());

	unsigned threadsPerBlock = 32;
	unsigned blockCount = (arraySize + (threadsPerBlock - 1)) / threadsPerBlock;
	unsigned* composites;
	hipError_t cudaStatus = hipMallocManaged(&composites, arraySize * sizeof(unsigned));

	sieve<<<blockCount, threadsPerBlock>>>(composites);

	hipDeviceSynchronize();

	auto end(std::chrono::high_resolution_clock::now());

	printResults(std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count(), composites);

	hipFree(composites);

    return 0;
}